
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <random>

#define N_RANGE 10000000
#define N_BINS 32

void gen_random_nums(int N, int* nums){
    // Generate N random integers.
    for (int i = 0; i < N; i++) {
        nums[i] = rand() % N_RANGE + 1;
    }
}

void bin_nums_CPU(int N, int *nums, int *bins){

    int bin;

    for (int i = 0; i < N; i++){ 
        bin = round(nums[i] / (N_RANGE/N_BINS));
        bins[bin]++;
        // printf("bin[%i] = %i\n", bin, bins[bin]);
    }
}

// __global__ void bin_nums_GPU()

int main(int argc, char* argv[]){
    
    srand(time(NULL));
    // Number of integers to generate
    int N = atoi(argv[1]); 

    int *nums;
    int *bins;


    // Allocation unified memory
    (int*)hipMallocManaged(&nums, N*sizeof(int));
    (int*)hipMallocManaged(&bins, N_BINS*sizeof(int));

    // device event objects
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // printf("generating numbers...");
    gen_random_nums(N, nums);
    bin_nums_CPU(N, nums, bins);

    // start timing all the device operations
    hipEventRecord(start, 0);

    

    

    for (int i = 0; i < N_BINS; i++) {
        printf("bin[%i] = %i\t", i, bins[i]);
        printf("\n");
    }

    // wait for the device to finish so that we see the message
    hipDeviceSynchronize();
    hipFree(nums);
    hipFree(bins);
    

    return 0;
}