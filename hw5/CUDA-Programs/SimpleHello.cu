
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ functions, or "kernels", execute on the device
__global__ void hello_kernel(void)
{
  printf("Hello, world from block %d, thread %d, on the device!\n", blockIdx.x, threadIdx.x);
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  hello_kernel<<<10,20>>>();

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}
